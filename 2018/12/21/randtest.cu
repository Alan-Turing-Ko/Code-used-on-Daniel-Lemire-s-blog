
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

#define OPENBSD       (1)
#define JAVA          (2)
#define DIVISIONLESS  (3)
#define DIVISIONLESSW (4)
  
#define RANDTEST_ITER            10
#define RANDTEST_THREADS         128
#define RANDTEST_DEFAULT_LEN     10000000
#define RANDTEST_DEFAULT_S       0x123456ULL
#define RANDTEST_DEFAULT_METHOD  OPENBSD

// Macro to catch CUDA errors in CUDA runtime calls
#define CUDA_SAFE_CALL(call)                                          \
do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

// Macro to catch CUDA errors in kernel launches
#define CHECK_LAUNCH_ERROR()                                          \
do {                                                                  \
    /* Check synchronous errors, i.e. pre-launch */                   \
    hipError_t err = hipGetLastError();                             \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
    /* Check asynchronous errors, i.e. kernel failed (ULF) */         \
    err = hipDeviceSynchronize();                                    \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString( err) );      \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

// A routine to give access to a high precision timer on most systems.
#if defined(_WIN32)
#if !defined(WIN32_LEAN_AND_MEAN)
#define WIN32_LEAN_AND_MEAN
#endif
#include <windows.h>
double second (void)
{
    LARGE_INTEGER t;
    static double oofreq;
    static int checkedForHighResTimer;
    static BOOL hasHighResTimer;

    if (!checkedForHighResTimer) {
        hasHighResTimer = QueryPerformanceFrequency (&t);
        oofreq = 1.0 / (double)t.QuadPart;
        checkedForHighResTimer = 1;
    }
    if (hasHighResTimer) {
        QueryPerformanceCounter (&t);
        return (double)t.QuadPart * oofreq;
    } else {
        return (double)GetTickCount() * 1.0e-3;
    }
}
#elif defined(__linux__) || defined(__APPLE__)
#include <stddef.h>
#include <sys/time.h>
double second (void)
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_sec + (double)tv.tv_usec * 1.0e-6;
}
#else
#error unsupported platform
#endif

/*
  Copyright (c) 2017, Norbert Juffa
  All rights reserved.

  Redistribution and use in source and binary forms, with or without 
  modification, are permitted provided that the following conditions
  are met:

  1. Redistributions of source code must retain the above copyright 
     notice, this list of conditions and the following disclaimer.

  2. Redistributions in binary form must reproduce the above copyright
     notice, this list of conditions and the following disclaimer in the
     documentation and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS 
  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT 
  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
  A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
  HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
  SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT 
  LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
  DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
  THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT 
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
// multiply two unsigned 64-bit integers into an unsigned 128-bit product
__device__ ulonglong2 umul64wide (uint64_t a, uint64_t b)
{
    ulonglong2 res;
#if (__CUDA_ARCH__ >= 200) && (__CUDA_ARCH__ < 500)
    asm ("{\n\t"
         ".reg .u32 r0, r1, r2, r3, alo, ahi, blo, bhi;\n\t"
         "mov.b64         {alo,ahi}, %2;\n\t"
         "mov.b64         {blo,bhi}, %3;\n\t"
         "mul.lo.u32      r0, alo, blo;\n\t"
         "mul.hi.u32      r1, alo, blo; \n\t"
         "mad.lo.cc.u32   r1, alo, bhi, r1;\n\t"
         "madc.hi.u32     r2, alo, bhi, 0;\n\t"
         "mad.lo.cc.u32   r1, ahi, blo, r1;\n\t"
         "madc.hi.cc.u32  r2, ahi, blo, r2;\n\t"
         "madc.hi.u32     r3, ahi, bhi, 0;\n\t"
         "mad.lo.cc.u32   r2, ahi, bhi, r2;\n\t"
         "addc.u32        r3, r3, 0;\n\t"
         "mov.b64         %0, {r0,r1};\n\t"  
         "mov.b64         %1, {r2,r3};\n\t"
         "}"
         : "=l"(res.x), "=l"(res.y)
         : "l"(a), "l"(b));
#elif __CUDA_ARCH__ >= 500
    asm ("{\n\t"
         ".reg .u32       alo, ahi, blo, bhi, r0, r1, r2, r3;\n\t"
         ".reg .u32       s0, s1, s2, s3, t0, t1, t2, t3;\n\t"
         ".reg .u16       a0, a1, a2, a3, b0, b1, b2, b3;\n\t"
         // split inputs into 16-bit chunks
         "mov.b64         {alo,ahi}, %2;\n\t"
         "mov.b64         {blo,bhi}, %3;\n\t"
         "mov.b32         {a0,a1}, alo;\n\t"
         "mov.b32         {a2,a3}, ahi;\n\t"
         "mov.b32         {b0,b1}, blo;\n\t"
         "mov.b32         {b2,b3}, bhi;\n\t"
         // first partial sum:
         // a3b3.wide  a1b3.wide  a0b2.wide  a0b0.wide
         //     0      a2b2.wide  a1b1.wide  
         //     0      a3b1.wide  a2b0.wide
         "mul.wide.u16    r0, a0, b0;\n\t"
         "mul.wide.u16    r1, a0, b2;\n\t"
         "mul.wide.u16    r2, a1, b3;\n\t"
         "mul.wide.u16    r3, a3, b3;\n\t"
         "mul.wide.u16    t1, a1, b1;\n\t"
         "mul.wide.u16    t2, a2, b2;\n\t"
         "add.cc.u32      r1, r1, t1;\n\t"
         "addc.cc.u32     r2, r2, t2;\n\t"
         "addc.u32        r3, r3, 0;\n\t"
         "mul.wide.u16    t1, a2, b0;\n\t"
         "mul.wide.u16    t2, a3, b1;\n\t"
         "add.cc.u32      r1, r1, t1;\n\t"
         "addc.cc.u32     r2, r2, t2;\n\t"
         "addc.u32        r3, r3, 0;\n\t"
         // second partial sum:
         // a2b3.wide  a0b3.wide  a0b1.wide
         // a3b2.wide  a1b2.wide  a1b0.wide 
         //     0      a2b1.wide
         //     0      a3b0.wide
         "mul.wide.u16    s0, a0, b1;\n\t"
         "mul.wide.u16    s1, a0, b3;\n\t"
         "mul.wide.u16    s2, a2, b3;\n\t"
         "mul.wide.u16    t1, a2, b1;\n\t"
         "add.cc.u32      s1, s1, t1;\n\t"
         "addc.u32        s2, s2, 0;\n\t"
         "mul.wide.u16    t1, a3, b0;\n\t"
         "add.cc.u32      s1, s1, t1;\n\t"
         "addc.u32        s2, s2, 0;\n\t"
         "mul.wide.u16    t0, a1, b0;\n\t"
         "mul.wide.u16    t1, a1, b2;\n\t"
         "mul.wide.u16    t2, a3, b2;\n\t"
         "add.cc.u32      s0, s0, t0;\n\t"
         "addc.cc.u32     s1, s1, t1;\n\t"
         "addc.cc.u32     s2, s2, t2;\n\t"
         "addc.u32        s3, 0, 0;\n\t"
         // offset second partial sum by 16 bits to the left
         "shf.l.clamp.b32 t3, s2, s3, 16;\n\t"
         "shf.l.clamp.b32 t2, s1, s2, 16;\n\t"
         "shf.l.clamp.b32 t1, s0, s1, 16;\n\t"
         "shf.l.clamp.b32 t0,  0, s0, 16;\n\t"
         // add first sum in r{0,1,2,3} to second sum in t{0,1,2,3}
         "add.cc.u32      r0, r0, t0;\n\t"
         "addc.cc.u32     r1, r1, t1;\n\t"
         "addc.cc.u32     r2, r2, t2;\n\t"
         "addc.u32        r3, r3, t3;\n\t"
         // pack outputs
         "mov.b64         %0, {r0,r1};\n\t"
         "mov.b64         %1, {r2,r3};\n\t"
         "}"
         : "=l"(res.x), "=l"(res.y)
         : "l"(a), "l"(b));
#elif __CUDA_ARCH__
#error unsupported __CUDA_ARCH__
#else // avoid warning
    res.x = 0;
    res.y = 0;
#endif
    return res;
}

/* state equations for Marsaglia's KISS64 PRNG */
#define MWC64 (kiss64->t = (kiss64->x << 58) + kiss64->c, \
               kiss64->c = (kiss64->x >> 6), kiss64->x += kiss64->t, \
               kiss64->c += (kiss64->x < kiss64->t), kiss64->x)
#define XSH64 (kiss64->y ^= (kiss64->y << 13), kiss64->y ^= (kiss64->y >> 17), \
               kiss64->y ^= (kiss64->y << 43))
#define CNG64 (kiss64->z = 6906969069ULL * kiss64->z + 1234567ULL)
#define KISS64 (MWC64 + XSH64 + CNG64)

/* KISS64 state */
typedef struct state_tag {
    uint64_t x, c, y, z, t;
} state;

__device__ uint64_t random64 (state *kiss64)
{
    return KISS64;
}

__device__ void seed_random64 (state *kiss64, int seed)
{
   kiss64->x = 1234567890987654321ULL;
   kiss64->c = 123456123456123456ULL;
   kiss64->y = 362436362436362436ULL;
   kiss64->z = 1066149217761810ULL + seed; /* cheesy "parallelization" */
   kiss64->t = 0LL;
   random64 (kiss64);
}

// returns value in [0,s)
__device__ uint64_t openbsd (state *kiss64, uint64_t s) 
{
    uint64_t t = (-s) % s;
    uint64_t x;
    do {
        x = random64 (kiss64);
    } while (x < t);
    return x % s;
}

// returns value in [0,s)
__device__ uint64_t java (state *kiss64, uint64_t s) 
{
    uint64_t x = random64 (kiss64);
    uint64_t r = x % s;
    while (x - r > UINT64_MAX - s + 1) {
        x = random64 (kiss64);
        r = x % s;
    }
    return r;
}

// returns value in [0,s)
__device__ uint64_t nearlydivisionless (state *kiss64, uint64_t s) 
{
    uint64_t x = random64 (kiss64);
    uint64_t h = __umul64hi (x, s);
    uint64_t l = x * s;
    if (l < s) {
        uint64_t t = -s % s;
        while (l < t) {
            x = random64 (kiss64);
            h =__umul64hi (x, s);
            l = x * s;
        }
    }
    return h;
}

// returns value in [0,s)
// like nearlydivisionless(), but using custom function for full product
__device__ uint64_t nearlydivisionless_w (state *kiss64, uint64_t s) 
{
    uint64_t x = random64 (kiss64);
    ulonglong2 m = umul64wide (x, s);
    uint64_t h = m.y;
    uint64_t l = m.x;
    if (l < s) {
        uint64_t t = -s % s;
        while (l < t) {
            x = random64 (kiss64);
            m = umul64wide (x, s);
            h = m.y;
            l = m.x;
        }
    }
    return h;
}

/* generate len 64-bit random numbers in [0,s] and store in random_number[] */
__global__ void randtest (uint64_t *randnum, int len, uint64_t s, int method)
{
    int stride = gridDim.x * blockDim.x;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    state kiss64 = {0};
    uint64_t num;

    seed_random64 (&kiss64, tid);

    for (int i = tid; i < len; i += stride) {
        switch (method) {
        case OPENBSD:
            num = openbsd (&kiss64, s);
            break;
        case JAVA:
            num = java (&kiss64, s);
            break;
        case DIVISIONLESS:
            num = nearlydivisionless (&kiss64, s);
            break;
        case DIVISIONLESSW:
            num = nearlydivisionless_w (&kiss64, s);
            break;
        default:
            printf ("*** bad method\n");
            break;
        }
        randnum[i] = num;
    }
}    

struct randtestOpts {
    uint64_t s;
    int len;
    int method;
};

int processArgs (int argc, char *argv[], struct randtestOpts *opts)
{
    int error = 0;
    memset (opts, 0, sizeof(*opts));
    while (argc) {
        if (*argv[0] == '-') {
            switch (*(argv[0]+1)) {
            case 'n':
                opts->len = atol(argv[0]+2);
                break;
            case 's':
                opts->s = atol(argv[0]+2);
                break;
            case 'm':
                opts->method =atol(argv[0]+2);
                if ((opts->method != OPENBSD) &&
                    (opts->method != JAVA) &&
                    (opts->method != DIVISIONLESS) &&
                    (opts->method != DIVISIONLESSW)) {
                    fprintf (stderr, "bad method: %s\n", argv[0]+2);
                    error++;
                }
                break;
            default:
                fprintf (stderr, "Unknown switch '%c%s'\n", '-', argv[0]+1);
                error++;
                break;
            }
        }
        argc--;
        argv++;
    }
    return error;
}
    
int main (int argc, char *argv[])
{
    double start, stop, elapsed, mintime;
    uint64_t *a=0, *d_a = 0;
    int errors;
    struct randtestOpts opts;

    errors = processArgs (argc, argv, &opts);
    if (errors) {
        return EXIT_FAILURE;
    }
    opts.len = (opts.len) ? opts.len : RANDTEST_DEFAULT_LEN;
    opts.s = (opts.s) ? opts.s : RANDTEST_DEFAULT_S;
    opts.method = (opts.method) ? opts.method : RANDTEST_DEFAULT_METHOD;

    /* Allocate memory on host */
    a = (uint64_t *)malloc(sizeof(a[0]) * opts.len);
    if (!a) {
        printf ("host allocation failed\n");
        return EXIT_FAILURE;
    }

    /* Allocate memory on device */
    CUDA_SAFE_CALL (hipMalloc((void**)&d_a, sizeof(d_a[0]) * opts.len));
    
    /* Initialize device memory */
    CUDA_SAFE_CALL (hipMemset(d_a, 0x00, sizeof(d_a[0]) * opts.len)); 

    /* Compute execution configuration */
    dim3 dimBlock(RANDTEST_THREADS);
    int threadBlocks = (opts.len + (dimBlock.x - 1)) / dimBlock.x;
    if (threadBlocks > 65520) threadBlocks = 65520;
    dim3 dimGrid(threadBlocks);
    
    printf ("randtest: operating on vectors of %d uint64_t\n", 
            opts.len);
    printf ("randtest: generating random numbers in [0,%llu]\n", opts.s);
    switch (opts.method) {
    case OPENBSD:
        printf ("randtest: range-restricting method is OPENBSD\n");
        break;
    case JAVA:
        printf ("randtest: range-restricting method is JAVA\n");
        break;
    case DIVISIONLESS:
        printf ("randtest: range-restricting method is DIVISIONLESS\n");
        break;
    case DIVISIONLESSW:
        printf ("randtest: range-restricting method is DIVISIONLESS_W\n");
        break;
    default:
        printf ("unsupported range-restricting method\n");
        break;
    }

    printf ("randtest: using %d threads per block, %d blocks\n", 
            dimBlock.x, dimGrid.x);

    mintime = fabs(log(0.0));
    for (int k = 0; k < RANDTEST_ITER; k++) {
        start = second();
        randtest<<<dimGrid,dimBlock>>>(d_a, opts.len, opts.s, opts.method);
        CHECK_LAUNCH_ERROR();
        stop = second();
        elapsed = stop - start;
        if (elapsed < mintime) mintime = elapsed;
    }
    printf ("randtest: mintime = %.3f msec\n",
            1.0e3 * mintime);

    /* Retrieve generated random numbers */
    CUDA_SAFE_CALL (hipMemcpy (a, d_a, sizeof(d_a[0]) * opts.len, hipMemcpyDeviceToHost));

    /* Check that generated num,bers are within desired range */
    for (int i = 0; i < opts.len; i++) {
        if (a [i] >= opts.s) {
            printf ("@ i=%d out of range [0, %llu]: %016llx\n", i, opts.s, a[i]);
        }
    }

    /* free memory on host and device */    
    CUDA_SAFE_CALL (hipFree(d_a));
    free (a);

    return EXIT_SUCCESS;
}
